#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<string>
#include <ctime>
using namespace std;

//�ϥΪ̩w�q
int NumofLight;        //�O���Ӽ�
int NumofInstruction; //�G�׫��O�Ӽ�
int NumofTd;          //�ؼ��I�Ӽ�

#define MaxNumofLight 12        //�O���Ӽ�
#define MaxNumofInstruction 16 //�G�׫��O�Ӽ�
#define MaxNumofTd 5          //�ؼ��I�Ӽ�

int NumofGroup;  //���s�ƥءA�W�[���ѼƯണ�ɮį�

float RepeatValue = -1;
float DifferentRange;
int NumofParticle;       //��NumofParticle�ոѧ@��PSO�����(�ɤl��) �A�C��particle�j�p��200
int NumofRedistribute;



#define OtherLight 0        //�B�~�������Ӷ�

#define ThreadX 8  ///�\��y��X��thread
#define ThreadY 8  ///�\��y��Y��thread

#define BlockX  128 ///�\��y��X��block 
#define BlockY  128 ///�\��y��Y��block

dim3 ThreadPerBlocks(ThreadX, ThreadY);//threads�bblock�����\��覡�A�y��X�PY
dim3 NumofBlocks(BlockX / ThreadPerBlocks.x, BlockY / ThreadPerBlocks.y);//block���\��覡�A�y��X�PY
//float OtherLightT[OtherLight]={0};        //�B�~�������ӫ�


//���ե�
int RunTime = 1;  //����h�֦�
float RealBest = -1;
#define in_cir_stable 2
#define in_cir_threshold 0.5


//�{���۩w�q
#define ImpossibleResume 99999
#define InitialW 0.9             //�D�ʫY�ơA�Ѥj��p
#define FinalW 0.4
#define InitialCOne 2.5          //������C�Y�ơA�Ѥj��p�A�]���j�M�����������
#define FinalCOne 1
#define InitialCTwo  1       //�s�骺C�Y�ơA�Ѥp��j�A�]���j�M���������s��
#define FinalCTwo 2.5

#define CircleofPSO 50
#define VelocityLimit 5
#define NumofGroupCircle 10



float Host_I[MaxNumofInstruction][2];   //(�ӯ�,�G��)
float* Host_Td;
float Host_KParameter[MaxNumofTd][MaxNumofLight + OtherLight];

int CheckT(float T, int MatrixRow, int i, int j, int k);
void PSO();
int RandomNumber(int MinValue, int MaxValue);
void* new2d(int h, int w, int size);
float similarity(int* A, int* B, int Length);
__device__ float device_similarity(int* A, int* B, int Length);
__device__ int device_RandomNumber(unsigned int thread_id, int MinValue, int MaxValue);

class Particle  //�C�Ӳɤl���򥻳]�w�A�ܰʽե����O����
{
private:
	int PBest[MaxNumofLight];  //(�Ĥ@���O���O,�ĤG���O���O,�ĤT���O���O,����̧C�ӯ�)
	float PBestFitness;
	int GBest[MaxNumofLight];
	float GBestFitness;
	int Position[MaxNumofLight];  //�O������A��U���O�����ե����O <----> Position   //�䤤�O���Ƥ]�Osolution�����סA�ĴX���O��ܲĴX��
	int Velocity[MaxNumofLight];             //�ե����O�W�ɩΤU�� <----> Velocity

public:
	Particle(int* PPointer, int* VPointer);
	Particle(){ PBestFitness = ImpossibleResume; }
	__host__  int* GetPBest(){ return PBest; }
	__host__  float GetPBestFitness(){ return PBestFitness; }
	__host__  void CheckPosition()
	{
		float Trow = 0;  //�ھڥثe�ɤl����m�Һ�X�Ӫ�T(�ӫ�)��
		for (int i = 0; i<NumofTd; i++)
		{
			for (int j = 0; j<(NumofLight); j++)
			{
				Trow += Host_KParameter[i][j] * Host_I[Position[j]][1];
			}
			if (Trow<Host_Td[i])
			{
				break;
			}
			Trow = 0;
			if (i == (NumofTd - 1))
			{
				ComputingFitness();
			}
		}
	}
	__host__  void ComputingFitness()
	{
		float Fitness = 0;
		for (int i = 0; i<NumofLight; i++)
		{
			Fitness += Host_I[Position[i]][0];
		}

		if (Fitness<PBestFitness || PBestFitness == ImpossibleResume)
		{
			for (int j = 0; j<NumofLight; j++)
			{
				PBest[j] = Position[j];
			}
			PBestFitness = Fitness;
		}
	}

	__device__  int* d_GetPBest(){ return PBest; }
	__device__  float d_GetPBestFitness(){ return PBestFitness; }
	__device__  void DeviceFunction(int* GBestPointer, float NewGBestFitness, float C0, float C1, float C2, float* I, float* Td, float* KParameter, int NumofLight, int NumofInstruction, int NumofTd, int ParticleNoinGroup)
	{
		//SetGBest
		GBestFitness = NewGBestFitness;

		//SetGBest & UpdateVelocity & Position 
		for (int i = 0; i<NumofLight; i++)//i�O����
		{
			//SetGBest
			GBest[i] = *(GBestPointer + i);

			//UpdateVelocity
			Velocity[i] = (C0*Velocity[i] + C1*(float)device_RandomNumber(ParticleNoinGroup, 0, 100) / 100 * (PBest[i] - Position[i]) + C2*(float)device_RandomNumber(ParticleNoinGroup,0, 100) / 100 * (GBest[i] - Position[i]));
			if (Velocity[i]<(-VelocityLimit))                //�t�פW�U��
			{
				Velocity[i] = (-VelocityLimit);
			}
			else if (Velocity[i]>VelocityLimit)                   //�t�פW�U��
			{
				Velocity[i] = VelocityLimit;
			}

			//UpdatePosition 
			Position[i] = Position[i] + Velocity[i];
			if (Position[i]<0)                                //�ե����O���W�U��
			{
				Position[i] = 0;
			}
			else if (Position[i]>(NumofInstruction - 1))             //�ե����O���W�U��
			{
				Position[i] = (NumofInstruction - 1);
			}
		}
		//CheckPosition
		float Trow = 0;  //�ھڥثe�ɤl����m�Һ�X�Ӫ�T(�ӫ�)��     

		for (int i = 0; i<NumofTd; i++)
		{
			for (int j = 0; j<(NumofLight + OtherLight); j++)
			{
				Trow += *(KParameter + i*(MaxNumofLight + OtherLight) + j) * *(I + 2 * Position[j] + 1);//KParameter*I;  
			}
			if (Trow<*(Td + i))
			{
				break;
			}
			Trow = 0;
			if (i == (NumofTd - 1))
			{
				//ComputingFitness
				float Fitness = 0;
				for (int i = 0; i<NumofLight; i++)
				{
					Fitness += *(I + 2 * Position[i] + 0);
				}

				if (Fitness<PBestFitness)
				{
					for (int j = 0; j<NumofLight; j++)
					{
						PBest[j] = Position[j];
					}
					PBestFitness = Fitness;
				}
				//ComputingFitnessEND
			}
		}
		//CheckPositionEND
	}

};

Particle::Particle(int* PPointer, int* VPointer)
{
	for (int i = 0; i<NumofLight; i++)
	{
		Position[i] = *(PPointer + i);
		Velocity[i] = *(VPointer + i);
		PBest[i] = Position[i];
	}
	PBestFitness = ImpossibleResume;
}//PSO���O����


__global__ void PSOKernel(Particle* Group, int* GBest, float* GBestFitness, int GroupCircle, float* I, float* Td, float* KParameter, int NumofLight, int NumofInstruction, int NumofTd, int NumofParticle)
{
	int CoordinatesX = blockIdx.x * blockDim.x + threadIdx.x;
	int CoordinatesY = blockIdx.y * blockDim.y + threadIdx.y;
	int ParticleNoinGroup = CoordinatesY*(ThreadX*BlockX) + CoordinatesX;//Y*(row size)+(X)
	if (ParticleNoinGroup < NumofParticle && CoordinatesX < (ThreadX*BlockX) && CoordinatesY < (ThreadY*BlockY))
	{
		float last_sim = -1;
		int stable_count = 0;
		for (int circle = 1; circle < CircleofPSO; circle++)
		{
			float C0 = InitialW - (InitialW - FinalW)*((circle + GroupCircle) / (NumofGroupCircle + CircleofPSO - 1));
			float C1 = InitialCOne - (InitialCOne - FinalCOne)*((circle + GroupCircle) / (NumofGroupCircle + CircleofPSO - 1));
			float C2 = InitialCTwo + (FinalCTwo - InitialCTwo)*((circle + GroupCircle) / (NumofGroupCircle + CircleofPSO - 1));

			(Group + ParticleNoinGroup)->DeviceFunction(GBest, *(GBestFitness), C0, C1, C2, I, Td, KParameter, NumofLight, NumofInstruction, NumofTd, ParticleNoinGroup);

			if (((Group + ParticleNoinGroup)->d_GetPBestFitness()) < *(GBestFitness))
			{
				for (int j = 0; j < NumofLight; j++)
				{
					*(GBest + j) = *(((Group + ParticleNoinGroup)->d_GetPBest()) + j);
				}
				*(GBestFitness) = (Group + ParticleNoinGroup)->d_GetPBestFitness();
			}
			
			//similarity
			float sim = 0;
			sim = device_similarity((Group + ParticleNoinGroup)->d_GetPBest(), (Group + ParticleNoinGroup)->d_GetPBest(), NumofLight);

			float temp_check = (sim - last_sim) / last_sim;
			if ((temp_check >= 0 && temp_check <= in_cir_threshold) || (temp_check < 0 && temp_check >= -in_cir_threshold))
			{
				stable_count++;
			}
			if (stable_count >= in_cir_stable)
			{
				break;
			}
			//circle--;
			last_sim = sim;
		}
	}
}


int main()
{
	/***********************Ū���ɮ�**********************/
	char FileInput[50];
	fstream fr;
	srand((unsigned)time(NULL));

	//Number of Particles
	fr.open("PSOParameter.txt", ios::in); //C:\\ITLab_cll\\LightIllumination\\LightIlluminationProgram\\PSOParameter.txt
	if (!fr)        //�p�G�}���ɮץ��ѡAfin��0�F���\�Afin��1
		cout << "Fail to open PSOParameter file" << endl;
	else
	{
		fr.getline(FileInput, sizeof(FileInput), ',');
		sscanf_s(FileInput, "%d", &NumofParticle);
		NumofRedistribute = NumofParticle / 2;
		fr.getline(FileInput, sizeof(FileInput), ',');
		sscanf_s(FileInput, "%f", &DifferentRange);
		fr.getline(FileInput, sizeof(FileInput), ',');
		sscanf_s(FileInput, "%d", &NumofGroup);
	}
	fr.close();

	//Light Instruction
	fr.open("LightInstruction.txt", ios::in);//C:\\ITLab_cll\\LightIllumination\\LightIlluminationProgram\\LightInstruction.txt
	if (!fr)        //�p�G�}���ɮץ��ѡAfin��0�F���\�Afin��1
		cout << "Fail to open file" << endl;
	else
	{
		fr.getline(FileInput, sizeof(FileInput), ',');//�o�ONumofInstruction
		sscanf(FileInput, "%d", &NumofInstruction);
		//Host_I = (float**)new2d(MaxNumofInstruction, 2, sizeof(float));
		for (int i = 0; i<NumofInstruction; i++)
		{
			for (int j = 0; j<2; j++)
			{
				fr.getline(FileInput, sizeof(FileInput), ',');
				sscanf(FileInput, "%f", &Host_I[i][j]);        //�r����Ʀr
				//cout << Host_I[i][j] << ",";
			}
			//cout<<endl;
		}
	}
	fr.close();

	//Td
	fr.open("Td.txt", ios::in);//C:\\ITLab_cll\\LightIllumination\\LightIlluminationProgram\\Td.txt
	if (!fr)        //�p�G�}���ɮץ��ѡAfin��0�F���\�Afin��1
		cout << "Fail to open file" << endl;
	else
	{
		fr.getline(FileInput, sizeof(FileInput), ',');//�o�ONumofTd
		sscanf(FileInput, "%d", &NumofTd);
		Host_Td = new float[NumofTd];
		for (int i = 0; i<NumofTd; i++)
		{
			fr.getline(FileInput, sizeof(FileInput), ',');
			sscanf(FileInput, "%f", &Host_Td[i]);        //�r����Ʀr
			//cout<<Host_Td[i]<<endl;
		}
	}
	fr.close();

	//KParameter
	fr.open("KParameter.txt", ios::in);//C:\\ITLab_cll\\LightIllumination\\LightIlluminationProgram\\KParameter.txt
	if (!fr)        //�p�G�}���ɮץ��ѡAfin��0�F���\�Afin��1
		cout << "Fail to open file" << endl;
	else
	{
		fr.getline(FileInput, sizeof(FileInput), ',');//�x�}��
		fr.getline(FileInput, sizeof(FileInput), ',');//�x�}�e
		sscanf(FileInput, "%d", &NumofLight);
		for (int i = 0; i<NumofTd; i++)
		{
			for (int j = 0; j<(NumofLight + OtherLight); j++)
			{
				fr.getline(FileInput, sizeof(FileInput), ',');
				sscanf(FileInput, "%f", &Host_KParameter[i][j]);        //�r����Ʀr
				//cout<<Host_KParameter[i][j]<<",";
			}
			//cout<<endl;
		}
	}
	fr.close();
	//���Ū������

	for (int i = 0; i<RunTime; i++)
	{
		PSO();
	}
	//system("pause");
	return 0;
}




void PSO()
{
	fstream fw;
	string filename;

	clock_t StartTime, EndTime;
	int* TotalGBest = new int[NumofLight];
	float TotalGBestFitness = ImpossibleResume;

	int** GBest = (int**)new2d(NumofGroup, MaxNumofLight, sizeof(int));
	float* GBestFitness = new float[NumofGroup];

	for (int i = 0; i<NumofGroup; i++)
	{
		GBestFitness[i] = ImpossibleResume;
	}

	int ProduceParticleNo = 0;

	Particle* GroupPointer;
	Particle** GroupInfo = (Particle**)new2d(NumofGroup, NumofParticle, sizeof(Particle));

	float total_ave_sim = 0;

	////////for CUDA//////////////////////

	//GPU�ǳƪŶ�
	float* I;   //(�ӯ�,�G��)
	float* Td;
	float* KParameter;

	size_t SizeofI = MaxNumofInstruction * 2 * sizeof(float);
	hipMalloc(&I, SizeofI);

	size_t SizeofTd = NumofTd*sizeof(float);
	hipMalloc(&Td, SizeofTd);

	size_t SizeofKParameter = MaxNumofTd*(MaxNumofLight + OtherLight)*sizeof(float);
	hipMalloc(&KParameter, SizeofKParameter);

	//�N��Ʋ���GPU
	hipMemcpy(I, Host_I, SizeofI, hipMemcpyHostToDevice);
	hipMemcpy(Td, Host_Td, SizeofTd, hipMemcpyHostToDevice);
	hipMemcpy(KParameter, Host_KParameter, SizeofKParameter, hipMemcpyHostToDevice);
	//���ʧ���

	int* d_GBest;
	Particle* d_Group;
	float* d_GBestFitness;

	size_t SizeofGroup = NumofParticle*sizeof(Particle);
	size_t SizeofGBestFitness = sizeof(float);
	size_t SizeofGBest = NumofLight*sizeof(int);


	hipMalloc(&d_Group, SizeofGroup);
	hipMalloc(&d_GBestFitness, SizeofGBestFitness);
	hipMalloc(&d_GBest, SizeofGBest);

	///////////////////////////////////////////////////////
	StartTime = clock(); //�}�l�ɶ�


	/********************************�H�����ͪ�l��*****************************************/
	int* RandomVelocity = new int[NumofLight];
	for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
	{
		ProduceParticleNo = 0;
		for (int i = 0; i<NumofParticle; i++)
		{
			int* j = new int[NumofLight];
			//�ɤl��l��m�H�����ͤ覡
			for (int h = 0; h<NumofLight; h++)
			{
				j[h] = RandomNumber(0, NumofInstruction - 1);
				RandomVelocity[h] = (RandomNumber(0, 1) ? -1 : 1)*RandomNumber(0, VelocityLimit);
			}
			GroupPointer = new Particle(j, RandomVelocity);
			GroupPointer->CheckPosition();
			if ((GroupPointer->GetPBestFitness())<GBestFitness[GroupNo])
			{
				GBestFitness[GroupNo] = GroupPointer->GetPBestFitness();
				for (int j = 0; j<NumofLight; j++)
				{
					GBest[GroupNo][j] = *(GroupPointer->GetPBest() + j);
				}
			}
			GroupInfo[GroupNo][ProduceParticleNo] = *GroupPointer;  //�h�����
			delete GroupPointer;                  //����Ŷ�
			ProduceParticleNo++;
		}
	}

	//��Ȳ��ͧ���
	//�P�ɡA�Ĥ@��PSO�����A�ѤUCircleofPSO-1��
	//�s�W����B�z�A�b�̥~�h
	/*******************************************************************************************************/
	for (int GroupCircle = 0; GroupCircle<NumofGroupCircle; GroupCircle++)
	{
		for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
		{
			//////////////CUDA
			hipMemcpy(d_Group, GroupInfo[GroupNo], SizeofGroup, hipMemcpyHostToDevice);
			hipMemcpy(d_GBestFitness, &GBestFitness[GroupNo], SizeofGBestFitness, hipMemcpyHostToDevice);
			hipMemcpy(d_GBest, GBest[GroupNo], SizeofGBest, hipMemcpyHostToDevice);


			PSOKernel << <NumofBlocks, ThreadPerBlocks >> >(d_Group, d_GBest, d_GBestFitness, GroupCircle, I, Td, KParameter, NumofLight, NumofInstruction, NumofTd, NumofParticle);


			hipMemcpy(GroupInfo[GroupNo], d_Group, SizeofGroup, hipMemcpyDeviceToHost);
			hipMemcpy(&GBestFitness[GroupNo], d_GBestFitness, SizeofGBestFitness, hipMemcpyDeviceToHost);
			hipMemcpy(GBest[GroupNo], d_GBest, SizeofGBest, hipMemcpyDeviceToHost);
			///////CODA over///////

			//similarity
			float sim_ave = 0;
			for (int i = 0; i < NumofParticle; i++)
			{
				sim_ave += similarity(GBest[GroupNo], GroupInfo[GroupNo][i].GetPBest(), NumofLight);
			}
			sim_ave = sim_ave / NumofParticle;

			//cout << "Group" << GroupNo + 1 << ", �{�b�������ۦ���:" << sim_ave << endl;
			total_ave_sim += sim_ave;
		}


		for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
		{
			if (GBestFitness[GroupNo]<TotalGBestFitness || TotalGBestFitness == ImpossibleResume)
			{
				for (int p = 0; p<NumofLight; p++)
				{
					TotalGBest[p] = GBest[GroupNo][p];
				}
				TotalGBestFitness = GBestFitness[GroupNo];
			}
		}

		//�{���D�X���Ѥwí�w
		if (RepeatValue == -1 && TotalGBest[NumofLight] != ImpossibleResume)
			RepeatValue = TotalGBest[NumofLight];
		else if (RepeatValue - TotalGBest[NumofLight] < TotalGBest[NumofLight] * DifferentRange && RepeatValue != -1)
		{
			cout << "�~�j�馸��: " << GroupCircle << endl;
			break;
		}

		//�����������s�A���s�զX
		int RandomCPU;
		int RandomParticle1, RandomParticle2;
		Particle TempMemory;
		for (int j = 0; j<NumofGroup; j++)
		{
			for (int i = 0; i<NumofRedistribute; i++)
			{
				RandomCPU = RandomNumber(0, NumofGroup - 1);
				while (RandomCPU == j)
				{
					RandomCPU = RandomNumber(0, NumofGroup - 1);
				}
				RandomParticle1 = RandomNumber(0, NumofParticle - 1);
				RandomParticle2 = RandomNumber(0, NumofParticle - 1);
				//
				TempMemory = GroupInfo[j][RandomParticle1];
				GroupInfo[j][RandomParticle1] = GroupInfo[RandomCPU][RandomParticle2];
				GroupInfo[RandomCPU][RandomParticle2] = TempMemory;
			}
		}
		//��������//

		for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
		{
			for (int i = 0; i<NumofLight; i++)
			{
				GBest[GroupNo][i] = TotalGBest[i];
			}
			GBestFitness[GroupNo] = TotalGBestFitness;
		}
	}
	EndTime = clock();//�����ɶ�


	/***PSO���浲���A�[�ݵ��G***
	if (TotalGBestFitness != ImpossibleResume)
	{
	//cout<<"PSO�j�M"<<endl
	cout << "���O:";                                                                                 //fw<<"PSO�j�M"<<endl<<"���O:";
	for (int i = 0; i<NumofLight; i++)
	{
	cout << TotalGBest[i] << ",";                                                                            //fw<<GBest[i];
	}
	cout << "�ӯ�:" << TotalGBestFitness << endl;                                                  //fw<<endl<<"�ӯ�:"<<GBest[NumofLight]<<endl;
	}
	else
	{
	cout << "�L��" << endl;
	}
	cout << "�O��:" << double(EndTime - StartTime) / CLOCKS_PER_SEC << "��" << endl << endl;
	/*****************************/

	hipFree(d_GBest);
	hipFree(d_Group);
	hipFree(d_GBestFitness);

	hipFree(I);
	hipFree(Td);
	hipFree(KParameter);

	/*****write file*****/
	filename = "The_Best_Solution.txt";//C:\\ITLab_cll\\LightIllumination\\LightIlluminationProgram\\The_Best_Solution.txt
	fw.open(filename, ios::out);//�}���ɮ�
	if (!fw){//�p�G�}���ɮץ��ѡAfw��0�F���\�Afw���D0
		cout << "Fail to open file: " << filename << endl;
	}
	//cout << "�O����O:" << endl;
	if (TotalGBestFitness != ImpossibleResume)
	{
		for (int i = 0; i<NumofLight; i++)
		{
			//cout << TotalGBest[i] << ", ";
			fw << TotalGBest[i] << endl;
		}
		//cout << endl;
		//cout << "�ӯ�:" << TotalGBestFitness << endl;
		fw << TotalGBestFitness << endl;
	}
	else
	{
		//cout << "�L��" << endl;
		fw << "�L��" << endl;
	}
	//cout << "�Ӯ�: " << double(EndTime - StartTime) / CLOCKS_PER_SEC << endl;
	fw << double(EndTime - StartTime) / CLOCKS_PER_SEC << endl;
	fw.close();//�����ɮ�

	//cout << "���饭�����X����50�ɪ��ۦ���: " << total_ave_sim / (NumofGroupCircle*NumofGroup) << endl;

	//����Ŷ�             //GroupInfo �D�ʺA�}�C�A���Φۦ�R��  //�{�b�O�ʺA�F
	//for (int i = 0; i < NumofGroup; i++)
	//{
	//for (int j = 0; j < NumofParticle; j++)
	//{
	delete GroupInfo;
	//}
	//}
	//
	//system("pause");
}



/****************************************Random Number**********************************************
�����Ʊ檺�̤j�ȻP�̤p�ȡA�N�ಣ�ͤ���o��ӭȤ����H���ơA�o���H���üƤ]�i�൥��̤j�ȡA�ε���̤p�ȡC
��üƭȬO�̷Ӯɶ��Ҳ��͡A�ҥH���ƩI�s�ɡA�|�������P���üƭ�(���)�C
***************************************************************************************************/
int RandomNumber(int MinValue, int MaxValue)   //Both MinValue and MaxValue are included
{
	int R = (rand() % (MaxValue - MinValue + 1)) + MinValue;
	return R;
}

__device__ int device_RandomNumber(unsigned int thread_id,int MinValue, int MaxValue)   //Both MinValue and MaxValue are included
{
	unsigned int seed = thread_id;
	hiprandState s;
	// seed a random number generator 
	hiprand_init(seed, 0, 0, &s);

	int R = ((int)hiprand_uniform(&s) % (MaxValue - MinValue + 1)) + MinValue;
	return R;
}

/**********************************************
�ʺA����2��array
/**********************************************/
void* new2d(int h, int w, int size)
{
	register int i;
	void **p;

	p = (void**)new char[h*sizeof(void*) + h*w*size];
	for (i = 0; i < h; i++)
	{
		p[i] = ((char *)(p + h)) + i*w*size;
	}

	return p;
}

/***********************************************
�p��ۦ���
/**********************************************/
float similarity(int* A, int* B, int Length)
{
	//for case 1
	float variation = 0, innerProduct = 0;
	float Similarity;

	//Euclidean distance(�V�p�V��)
	for (int i = 0; i < Length; i++)
	{
		innerProduct += pow(A[i], 2);
		variation += pow(A[i] - B[i], 2);
	}
	Similarity = sqrt(variation) / sqrt(innerProduct);

	//cout <<"variation:"<< fixed << setprecision(5) << Similarity << ", ";
	return Similarity;
}


__device__ float device_similarity(int* A, int* B, int Length)
{
	//for case 1
	float variation = 0, innerProduct = 0;
	float Similarity;

	//Euclidean distance(�V�p�V��)
	for (int i = 0; i < Length; i++)
	{
		innerProduct += A[i] * A[i];
		variation += (A[i] - B[i]) * (A[i] - B[i]);
	}
	Similarity = sqrt(variation) / sqrt(innerProduct);

	//cout <<"variation:"<< fixed << setprecision(5) << Similarity << ", ";
	return Similarity;
}