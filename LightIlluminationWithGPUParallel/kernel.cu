
#include "hip/hip_runtime.h"


#include <stdio.h>
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<string>
#include <ctime>
using namespace std;

//�ϥΪ̩w�q
#define NumofLight 12        //�O���Ӽ�
#define NumofInstruction 16 //�G�׫��O�Ӽ�
#define NumofTd 5          //�ؼ��I�Ӽ�

#define NumofGroup 10  //���s�ƥءA�W�[���ѼƯണ�ɮį�


#define NumofParticle 100       //��NumofParticle�ոѧ@��PSO�����(�ɤl��) �A�C��particle�j�p��200
#define NumofRedistribute 50



#define OtherLight 0        //�B�~�������Ӷ�

#define ThreadX 4  ///�\��y��X��thread
#define ThreadY 4  ///�\��y��Y��thread

#define BlockX  100 ///�\��y��X��block 
#define BlockY  100 ///�\��y��Y��block

dim3 ThreadPerBlocks(ThreadX, ThreadY);//threads�bblock�����\��覡�A�y��X�PY
dim3 NumofBlocks(BlockX / ThreadPerBlocks.x, BlockY / ThreadPerBlocks.y);//block���\��覡�A�y��X�PY
//float OtherLightT[OtherLight]={0};        //�B�~�������ӫ�


//���ե�
int RunTime = 1;  //�]�h�֦�
float RealBest = -1;//246.34;


//�{���۩w�q
#define ImpossibleResume 99999
#define InitialW 0.9             //�D�ʫY�ơA�Ѥj��p
#define FinalW 0.4
#define InitialCOne 2.5          //������C�Y�ơA�Ѥj��p�A�]���j�M�����������
#define FinalCOne 1
#define InitialCTwo  1       //�s�骺C�Y�ơA�Ѥp��j�A�]���j�M���������s��
#define FinalCTwo 2.5

#define CircleofPSO 50
#define VelocityLimit 5
#define NumofGroupCircle 10

float RepeatValue = -1;
float DifferentRange = 0.0005;

//__shared__ int CircleofPSO;  
//__device__ int VelocityLimit;
//__shared__ int NumofGroupCircle;

/*
void Parameter()
{
CircleofPSO=NumofParticle*(0.02);      //PSO���檺�`�����ơA�ɤl�ƪ�2%
if(CircleofPSO<10)   //�������ƹL�ַ|�y���L��
CircleofPSO=10;

VelocityLimit=NumofInstruction*(0.5);  //���ʳt�פW���A���ʪŶ���50%
if(VelocityLimit<3)   //���ʭ�����Y�A�|�ɭP���鰱���
VelocityLimit=3;

NumofGroupCircle=NumofGroup;    //�U�B�z������y�һݡA�����B�z���Ӽ�
}
*/
//
//__shared__ float I[NumofInstruction][2];   //(�ӯ�,�G��)    //__shared__���O�A�bdeviceŪ���Ȯɷ|�y�����~
//__shared__ float Td[NumofTd];
//__shared__ float KParameter[NumofTd][NumofLight+OtherLight];


float** Host_I;   //(�ӯ�,�G��)
float* Host_Td;
float** Host_KParameter;

int CheckT(float T, int MatrixRow, int i, int j, int k);
float PSO();
int RandomNumber(int MinValue, int MaxValue);
void* new2d(int h, int w, int size);

class Particle  //�C�Ӳɤl���򥻳]�w�A�ܰʽե����O����
{
public:
	int* PBest;  //(�Ĥ@���O���O,�ĤG���O���O,�ĤT���O���O,����̧C�ӯ�)
	float PBestFitness;
	int GBest[NumofLight];
	float GBestFitness;
	int Position[NumofLight];  //�O������A��U���O�����ե����O <----> Position   //�䤤�O���Ƥ]�Osolution�����סA�ĴX���O��ܲĴX��
	int Velocity[NumofLight];             //�ե����O�W�ɩΤU�� <----> Velocity

public:
	Particle(int* PPointer, int* VPointer);
	Particle(){ PBestFitness = ImpossibleResume; }
	__host__  int* GetPBest(){ return PBest; }
	__host__  float GetPBestFitness(){ return PBestFitness; }
	__host__  void CheckPosition()
	{
		float Trow = 0;  //�ھڥثe�ɤl����m�Һ�X�Ӫ�T(�ӫ�)��
		for (int i = 0; i<NumofTd; i++)
		{
			for (int j = 0; j<(NumofLight); j++)
			{
				Trow += Host_KParameter[i][j] * Host_I[Position[j]][1];
			}
			if (Trow<Host_Td[i])
			{
				break;
			}
			Trow = 0;
			if (i == (NumofTd - 1))
			{
				ComputingFitness();
			}
		}
	}
	__host__  void ComputingFitness()
	{
		float Fitness = 0;
		for (int i = 0; i<NumofLight; i++)
		{
			Fitness += Host_I[Position[i]][0];
		}

		if (Fitness<PBestFitness || PBestFitness == ImpossibleResume)
		{
			for (int j = 0; j<NumofLight; j++)
			{
				PBest[j] = Position[j];
			}
			PBestFitness = Fitness;
		}
	}
	__host__  void SetGBest(int* GBestPointer, float NewGBestFitness)
	{
		for (int i = 0; i<NumofLight; i++)
		{
			GBest[i] = *(GBestPointer + i);
		}
		GBestFitness = NewGBestFitness;
	}

	__device__  int* d_GetPBest(){ return PBest; }
	__device__  float d_GetPBestFitness(){ return PBestFitness; }
	/***********
	__device__  void SetGBest(int* GBestPointer,float NewGBestFitness)
	{
	for(int i=0;i<NumofLight;i++)
	{
	GBest[i]=*(GBestPointer+i);
	}
	GBestFitness=NewGBestFitness;
	}
	__device__  void UpdatePosition()
	{
	for(int i=0;i<NumofLight;i++)//i�O����
	{
	Position[i]=Position[i]+Velocity[i];
	if(Position[i]<0)                                //�ե����O���W�U��
	{
	Position[i]=0;
	}
	else if(Position[i]>(NumofInstruction-1))             //�ե����O���W�U��
	{
	Position[i]=(NumofInstruction-1);
	}
	}
	}
	__device__  void UpdateVelocity(float C0,float C1,float C2)
	{
	for(int i=0;i<NumofLight;i++)//i�O����
	{
	Velocity[i]=(C0*Velocity[i]+C1*(PBest[i]-Position[i])+C2*(GBest[i]-Position[i]));
	if(Velocity[i]<(-VelocityLimit))                //�t�פW�U��
	{
	Velocity[i]=(-VelocityLimit);
	}
	if(Velocity[i]>VelocityLimit)                   //�t�פW�U��
	{
	Velocity[i]=VelocityLimit;
	}
	}
	}
	__device__  void d_CheckPosition(float* I,float* Td,float* KParameter)  //�s�b���D
	{
	float Trow=0;  //�ھڥثe�ɤl����m�Һ�X�Ӫ�T(�ӫ�)��

	for(int i=0;i<NumofTd;i++)
	{
	for(int j=0;j<(NumofLight+OtherLight);j++)
	{
	Trow+=*(KParameter+i*(NumofLight+OtherLight)+j) * *(I+2*Position[j]+1);//KParameter*I;
	}
	if(Trow<*(Td+i))
	{
	break;
	}
	Trow=0;
	if(i==(NumofTd-1))
	{
	d_ComputingFitness(I);
	}
	}
	}
	__device__  void d_ComputingFitness(float* I)
	{
	float Fitness=0;
	for(int i=0;i<NumofLight;i++)
	{
	Fitness += *(I+2*Position[i]+0);
	}

	if(Fitness<PBestFitness)
	{
	for(int j=0;j<NumofLight;j++)
	{
	PBest[j] = Position[j];
	}
	PBestFitness=Fitness;
	}
	}
	********/
	__device__  void DeviceFunction(int* GBestPointer, float NewGBestFitness, float C0, float C1, float C2, float* I, float* Td, float* KParameter)
	{
		//SetGBest
		GBestFitness = NewGBestFitness;

		//SetGBest & UpdateVelocity & Position 
		for (int i = 0; i<NumofLight; i++)//i�O����
		{
			//SetGBest
			GBest[i] = *(GBestPointer + i);

			//UpdateVelocity
			Velocity[i] = (C0*Velocity[i] + C1*(PBest[i] - Position[i]) + C2*(GBest[i] - Position[i]));
			if (Velocity[i]<(-VelocityLimit))                //�t�פW�U��
			{
				Velocity[i] = (-VelocityLimit);
			}
			else if (Velocity[i]>VelocityLimit)                   //�t�פW�U��
			{
				Velocity[i] = VelocityLimit;
			}

			//UpdatePosition 
			Position[i] = Position[i] + Velocity[i];
			if (Position[i]<0)                                //�ե����O���W�U��
			{
				Position[i] = 0;
			}
			else if (Position[i]>(NumofInstruction - 1))             //�ե����O���W�U��
			{
				Position[i] = (NumofInstruction - 1);
			}
		}
		//CheckPosition
		float Trow = 0;  //�ھڥثe�ɤl����m�Һ�X�Ӫ�T(�ӫ�)��     

		for (int i = 0; i<NumofTd; i++)
		{
			for (int j = 0; j<(NumofLight + OtherLight); j++)
			{
				Trow += *(KParameter + i*(NumofLight + OtherLight) + j) * *(I + 2 * Position[j] + 1);//KParameter*I;  
			}
			if (Trow<*(Td + i))
			{
				break;
			}
			Trow = 0;
			if (i == (NumofTd - 1))
			{
				//ComputingFitness
				float Fitness = 0;
				for (int i = 0; i<NumofLight; i++)
				{
					Fitness += *(I + 2 * Position[i] + 0);
				}

				if (Fitness<PBestFitness)
				{
					for (int j = 0; j<NumofLight; j++)
					{
						PBest[j] = Position[j];
					}
					PBestFitness = Fitness;
				}
				//ComputingFitnessEND
			}
		}
		//CheckPositionEND
	}
	//test
	__device__  void testSetPBest(int Value)
	{
		for (int i = 0; i<NumofLight; i++)
		{
			PBest[i] = Value;
		}
	}
};

Particle::Particle(int* PPointer, int* VPointer)
{
	PBest = new int[NumofLight];
	for (int i = 0; i<NumofLight; i++)
	{
		Position[i] = *(PPointer + i);
		Velocity[i] = *(VPointer + i);
		PBest[i] = Position[i];
	}
	PBestFitness = ImpossibleResume;
}//PSO���O����


__global__ void PSOKernel(Particle* Group, int* GBest, float* GBestFitness, int GroupCircle, float* I, float* Td, float* KParameter)
{
	int CoordinatesX = blockIdx.x * blockDim.x + threadIdx.x;
	int CoordinatesY = blockIdx.y * blockDim.y + threadIdx.y;
	int ParticleNoinGroup = CoordinatesY*(ThreadX*BlockX) + CoordinatesX;//Y*(row size)+(X)
	if (ParticleNoinGroup < NumofParticle && CoordinatesX < (ThreadX*BlockX) && CoordinatesY < (ThreadY*BlockY))
		for (int circle = 1; circle < CircleofPSO; circle++)
		{
		float C0 = InitialW - (InitialW - FinalW)*((circle + GroupCircle) / (NumofGroupCircle + CircleofPSO - 1));
		float C1 = InitialCOne - (InitialCOne - FinalCOne)*((circle + GroupCircle) / (NumofGroupCircle + CircleofPSO - 1));
		float C2 = InitialCTwo + (FinalCTwo - InitialCTwo)*((circle + GroupCircle) / (NumofGroupCircle + CircleofPSO - 1));
		/****
		//��sGBest
		(Group+ParticleNoinGroup)->SetGBest(GBest,*(GBestFitness));
		//��s�t��
		(Group+ParticleNoinGroup)->UpdateVelocity(C0,C1,C2);
		//��s��m
		(Group+ParticleNoinGroup)->UpdatePosition();
		//�ˬd�X��
		(Group+ParticleNoinGroup)->d_CheckPosition(I,Td,KParameter);
		/*****/
		(Group + ParticleNoinGroup)->DeviceFunction(GBest, *(GBestFitness), C0, C1, C2, I, Td, KParameter);
		/*****/
		//(Group+ParticleNoinGroup)->DelayFunction(); 
		/****
		__syncthreads();

		for(int e=0;e<NumofParticle;e++)
		{
		if(((Group+e)->d_GetPBestFitness())< *(GBestFitness) )
		{
		for(int j=0;j<NumofLight;j++)
		{
		*(GBest+j)=*(((Group+e)->d_GetPBest())+j);
		}
		*(GBestFitness)=(Group+e)->d_GetPBestFitness();
		}
		}
		/****/
		if (((Group + ParticleNoinGroup)->d_GetPBestFitness()) < *(GBestFitness))
		{
			for (int j = 0; j < NumofLight; j++)
			{
				*(GBest + j) = *(((Group + ParticleNoinGroup)->d_GetPBest()) + j);
			}
			*(GBestFitness) = (Group + ParticleNoinGroup)->d_GetPBestFitness();
		}
		/****/
		}
	if (ParticleNoinGroup == 0)  //for test
	{
		(Group + ParticleNoinGroup)->testSetPBest(-1);
	}
}


int main()
{
	/***********************Ū���ɮ�**********************/
	char FileInput[50];
	fstream fr;
	srand((unsigned)time(NULL));
	//Light Instruction
	fr.open("LightInstruction.txt", ios::in);
	if (!fr)        //�p�G�}���ɮץ��ѡAfin��0�F���\�Afin��1
		cout << "Fail to open file" << endl;
	else
	{
		fr.getline(FileInput, sizeof(FileInput), ',');//�o�ONumofInstruction
		Host_I = (float **)new2d(NumofInstruction, 2, sizeof(float));
		for (int i = 0; i<NumofInstruction; i++)
		{
			for (int j = 0; j<2; j++)
			{
				fr.getline(FileInput, sizeof(FileInput), ',');
				sscanf(FileInput, "%f", &Host_I[i][j]);        //�r����Ʀr
			}
		}
	}
	fr.close();

	//Td
	fr.open("Td.txt", ios::in);
	if (!fr)        //�p�G�}���ɮץ��ѡAfin��0�F���\�Afin��1
		cout << "Fail to open file" << endl;
	else
	{
		fr.getline(FileInput, sizeof(FileInput), ',');//�o�ONumofTd
		Host_Td = new float[NumofTd];
		for (int i = 0; i<NumofTd; i++)
		{
			fr.getline(FileInput, sizeof(FileInput), ',');
			sscanf(FileInput, "%f", &Host_Td[i]);        //�r����Ʀr
		}
	}
	fr.close();

	//KParameter
	fr.open("KParameter.txt", ios::in);
	if (!fr)        //�p�G�}���ɮץ��ѡAfin��0�F���\�Afin��1
		cout << "Fail to open file" << endl;
	else
	{
		fr.getline(FileInput, sizeof(FileInput), ',');//�x�}��
		fr.getline(FileInput, sizeof(FileInput), ',');//�x�}�e
		Host_KParameter = (float **)new2d(NumofTd, NumofLight + OtherLight, sizeof(float));
		for (int i = 0; i<NumofTd; i++)
		{
			for (int j = 0; j<(NumofLight + OtherLight); j++)
			{
				fr.getline(FileInput, sizeof(FileInput), ',');
				sscanf(FileInput, "%f", &Host_KParameter[i][j]);        //�r����Ʀr
			}
		}
	}
	fr.close();
	//���Ū������

	float PSOResult;
	int NumberofBest = 0;
	int Numberof5 = 0;
	float ResultTime = 0;
	int Others = 0;
	for (int i = 0; i<RunTime; i++)
	{
		PSOResult = PSO();
		if (PSOResult<RealBest + 0.001)
			NumberofBest++;
		else if (PSOResult<(RealBest + RealBest*0.05) && PSOResult>RealBest)
			Numberof5++;
		else
			Others++;
		//if(i!=0) 
		//ResultTime+=PSOResult;
	}
	/*********************���ק�PSO�禡���^�ǭ�************************/
	cout << "Best:" << NumberofBest << "��" << endl;
	cout << "5%:" << Numberof5 << "��" << endl;
	cout << "Others:" << Others << "��" << endl;
	//cout<<"�����O��"<< ResultTime/RunTime<<"��"<<endl;

	/***����Ŷ�***/
	delete[] Host_I;
	delete[] Host_Td;
	delete[] Host_KParameter;

	system("pause");
	return 0;
}




float PSO()
{
	fstream fw;
	string filename;

	clock_t StartTime, EndTime;
	int* TotalGBest=new int[NumofLight];
	float TotalGBestFitness = ImpossibleResume;

	int** GBest=(int**)new2d(NumofGroup,NumofLight,sizeof(int));
	float* GBestFitness=new float[NumofGroup];
	for (int i = 0; i<NumofGroup; i++)
	{
		GBestFitness[i] = ImpossibleResume;
	}

	int ProduceParticleNo = 0;

	Particle* GroupPointer;
	Particle** GroupInfo=(Particle**)new2d(NumofGroup,NumofParticle,sizeof(Particle));

	/*********���������t�ɨϥ�************
	Particle TempGroup[NumofGroup][NumofParticle];
	int TempGroupNo[NumofGroup];  //�p��洫�ɤl�Ϊ��Ȧs�s�O�_�w��
	for (int i = 0; i<NumofGroup; i++)
	{
	TempGroupNo[i] = 0;
	}
	/**************************************/
	////////for CUDA//////////////////////

	//GPU�ǳƪŶ�
	float* I;   //(�ӯ�,�G��)
	float* Td;
	float* KParameter;

	size_t SizeofI = NumofInstruction * 2 * sizeof(float);
	hipMalloc(&I, SizeofI);

	size_t SizeofTd = NumofTd*sizeof(float);
	hipMalloc(&Td, SizeofTd);

	size_t SizeofKParameter = NumofTd*(NumofLight + OtherLight)*sizeof(float);
	hipMalloc(&KParameter, SizeofKParameter);

	//�N��Ʋ���GPU
	hipMemcpy(I, Host_I + NumofInstruction, SizeofI, hipMemcpyHostToDevice);
	hipMemcpy(Td, Host_Td, SizeofTd, hipMemcpyHostToDevice);
	hipMemcpy(KParameter, Host_KParameter + NumofTd, SizeofKParameter, hipMemcpyHostToDevice);
	//���ʧ���

	int* d_GBest;
	Particle* d_Group;
	float* d_GBestFitness;

	size_t SizeofGroup = NumofParticle*sizeof(Particle);
	size_t SizeofGBestFitness = sizeof(float);
	size_t SizeofGBest = NumofLight*sizeof(int);

	size_t SizeofPBest = sizeof(int[NumofLight]);


	hipMalloc(&d_Group, SizeofGroup);
	
	for (int i = 0; i < NumofParticle; i++)
	{
		hipMalloc(&d_Group[i].PBest, SizeofPBest);    //CUDA�S��PBest�Ŷ��s����
	}

	hipMalloc(&d_GBestFitness, SizeofGBestFitness);
	hipMalloc(&d_GBest, SizeofGBest);

	///////////////////////////////////////////////////////
	StartTime = clock(); //�}�l�ɶ�


	/********************************�H�����ͪ�l��*****************************************/
	int* RandomVelocity=new int[NumofLight];
	for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
	{
		ProduceParticleNo = 0;
		for (int i = 0; i<NumofParticle; i++)
		{
			int* j=new int[NumofLight];
			//�ɤl��l��m�H�����ͤ覡
			for (int h = 0; h<NumofLight; h++)
			{
				j[h] = RandomNumber(0, NumofInstruction - 1);
				RandomVelocity[h] = (RandomNumber(0, 1) ? -1 : 1)*RandomNumber(0, VelocityLimit);
			}
			GroupPointer = new Particle(j, RandomVelocity);
			GroupPointer->CheckPosition();
			if ((GroupPointer->GetPBestFitness())<GBestFitness[GroupNo])
			{
				GBestFitness[GroupNo] = GroupPointer->GetPBestFitness();
				for (int j = 0; j<NumofLight; j++)
				{
					GBest[GroupNo][j] = *(GroupPointer->GetPBest() + j);
				}
			}
			GroupInfo[GroupNo][ProduceParticleNo] = *GroupPointer;  //�h�����
			delete GroupPointer;                  //����Ŷ�
			ProduceParticleNo++;
		}
	}
	//��l��m���ͧ���

	/*****��z�ɤl�b�O���餤����m(�[�t�N��ƶǤJGPU���t��)*****
	for(int GroupNo=0;GroupNo<NumofGroup;GroupNo++)
	{
	for(int i=1;i<NumofParticle;i++)
	{
	GroupInfo[GroupNo][i]=*Group[GroupNo][i];  //�h�����
	delete Group[GroupNo][i];                  //����Ŷ�
	}
	}
	/*****��z����*****/

	//��Ȳ��ͧ���
	//�P�ɡA�Ĥ@��PSO�����A�ѤUCircleofPSO-1��
	//�s�W����B�z�A�b�̥~�h
	/*******************************************************************************************************/
	for (int GroupClicle = 0; GroupClicle<NumofGroupCircle; GroupClicle++)
	{
		cout << "�e:";
		for (int i = 0; i < NumofLight; i++)
		{
			cout << GroupInfo[0][0].PBest[i] << ",";
		}cout << GroupInfo[0][0].PBestFitness << endl;
		for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
		{
			//////////////CUDA
			for (int i = 0; i < NumofParticle;i++)
			{
				hipMemcpy(d_Group[i].GBest, GroupInfo[GroupNo][i].GBest, sizeof(int[NumofLight]), hipMemcpyHostToDevice);
				hipMemcpy(&d_Group[i].GBestFitness, &GroupInfo[GroupNo][i].GBestFitness, sizeof(float), hipMemcpyHostToDevice);
				//cudaMemcpy(d_Group[i].PBest, GroupInfo[GroupNo][i].PBest, sizeof(int[NumofLight]), cudaMemcpyHostToDevice);  
				
				hipMemcpy(d_Group[i].PBest, GroupInfo[GroupNo][i].PBest, sizeof(int[NumofLight]), hipMemcpyHostToDevice);

				hipMemcpy(&d_Group[i].PBestFitness, &GroupInfo[GroupNo][i].PBestFitness, sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(d_Group[i].Position, GroupInfo[GroupNo][i].Position, sizeof(int[NumofLight]), hipMemcpyHostToDevice);
				hipMemcpy(d_Group[i].Velocity, GroupInfo[GroupNo][i].Velocity, sizeof(int[NumofLight]), hipMemcpyHostToDevice);
			}
			//cudaMemcpy(d_Group, GroupInfo[GroupNo], SizeofGroup, cudaMemcpyHostToDevice);
			hipMemcpy(d_GBestFitness, &GBestFitness[GroupNo], SizeofGBestFitness, hipMemcpyHostToDevice);
			hipMemcpy(d_GBest, GBest[GroupNo], SizeofGBest, hipMemcpyHostToDevice);


			PSOKernel <<< NumofBlocks, ThreadPerBlocks >>>(d_Group, d_GBest, d_GBestFitness, GroupClicle, I, Td, KParameter);


			for (int i = 0; i < NumofParticle; i++)
			{
				hipMemcpy(d_Group[i].GBest, GroupInfo[GroupNo][i].GBest, sizeof(int[NumofLight]), hipMemcpyDeviceToHost);
				hipMemcpy(&d_Group[i].GBestFitness, &GroupInfo[GroupNo][i].GBestFitness, sizeof(float), hipMemcpyDeviceToHost);

				hipMemcpy(d_Group[i].PBest, GroupInfo[GroupNo][i].PBest, sizeof(int[NumofLight]), hipMemcpyDeviceToHost);

				hipMemcpy(&d_Group[i].PBestFitness, &GroupInfo[GroupNo][i].PBestFitness, sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(d_Group[i].Position, GroupInfo[GroupNo][i].Position, sizeof(int[NumofLight]), hipMemcpyDeviceToHost);
				hipMemcpy(d_Group[i].Velocity, GroupInfo[GroupNo][i].Velocity, sizeof(int[NumofLight]), hipMemcpyDeviceToHost);
			}
			//cudaMemcpy(GroupInfo[GroupNo], d_Group, SizeofGroup, cudaMemcpyDeviceToHost);
			hipMemcpy(&GBestFitness[GroupNo], d_GBestFitness, SizeofGBestFitness, hipMemcpyDeviceToHost);
			hipMemcpy(GBest[GroupNo], d_GBest, SizeofGBest, hipMemcpyDeviceToHost);
			///////CODA over///////
		}
		cout << "��:";
		for (int i = 0; i < NumofLight; i++)
		{
			cout << GroupInfo[0][0].PBest[i] << ",";
		}cout << GroupInfo[0][0].PBestFitness << endl;


		for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
		{
			if (GBestFitness[GroupNo]<TotalGBestFitness || TotalGBestFitness == ImpossibleResume)
			{
				for (int p = 0; p<NumofLight; p++)
				{
					TotalGBest[p] = GBest[GroupNo][p];
				}
				TotalGBestFitness = GBestFitness[GroupNo];
			}
		}
		/*
		//�����覡
		//��ҭ���
		int RedistributeList[NumofGroup];
		int RedistributeParticleList[NumofGroup][NumofRedistribute];
		int RandomGroup;
		int RandomParticle;
		Particle TempMemory0[NumofRedistribute];
		Particle TempMemory1[NumofRedistribute];

		//�إ߸s��list
		for(int i=0;i<NumofGroup;i++)
		{
		RandomGroup=RandomNumber(0,NumofGroup-1);
		for(int j=0;j<i;j++)
		{
		if(RedistributeList[j]==RandomGroup)
		{
		RandomGroup=RandomNumber(0,NumofGroup-1);
		j=0;
		}
		}
		RedistributeList[i]=RandomGroup;
		}

		//��ܭn�洫���ɤl�A�إ߹�Ӫ�
		for(int k=0;k<NumofGroup;k++)
		{
		for(int i=0;i<NumofRedistribute;i++)
		{
		RandomParticle=RandomNumber(0,NumofParticle-1);
		for(int j=0;j<i;j++)
		{
		if(RedistributeParticleList[k][j]==RandomParticle)
		{
		RandomParticle=RandomNumber(0,NumofParticle-1);
		j=0;
		}
		}
		RedistributeParticleList[k][i]=RandomParticle;
		}
		}

		//�}�l�洫
		for(int i=0,k=0;k<NumofGroup;k++) //i�O�Ҧb���s,k�O�ĴX���@�s�������洫
		{
		if(k==0)
		{
		for(int j=0;j<NumofRedistribute;j++)
		{
		TempMemory0[j] = GroupInfo[i][ RedistributeParticleList[i][j] ];
		}
		}
		if((k%2)==0)
		{
		for(int j=0;j<NumofRedistribute;j++)
		{
		TempMemory1[j] = GroupInfo[ RedistributeList[i] ][ RedistributeParticleList[ RedistributeList[i] ][j] ];
		GroupInfo[ RedistributeList[i] ][ RedistributeParticleList[ RedistributeList[i] ][j] ] = TempMemory0[j];
		}
		}
		else
		{
		for(int j=0;j<NumofRedistribute;j++)
		{
		TempMemory0[j] = GroupInfo[ RedistributeList[i] ][ RedistributeParticleList[ RedistributeList[i] ][j] ];
		GroupInfo[ RedistributeList[i] ][ RedistributeParticleList[ RedistributeList[i] ][j] ] = TempMemory1[j];
		}
		}
		i=RedistributeList[i];
		}

		/********************************/
		/************��������************
		int RandomGroup;
		for (int i = 0; i<NumofGroup; i++)
		{
		for (int j = 0; j<NumofParticle; j++)
		{
		RandomGroup = RandomNumber(0, NumofGroup - 1);
		while (TempGroupNo[RandomGroup] == NumofParticle)//�Ȧs�s�w��
		{
		RandomGroup = RandomNumber(0, NumofGroup - 1);
		}
		TempGroup[RandomGroup][TempGroupNo[RandomGroup]++] = GroupInfo[i][j];
		}
		}
		for (int i = 0; i<NumofGroup; i++)
		{
		for (int j = 0; j<NumofParticle; j++)
		{
		GroupInfo[i][j] = TempGroup[i][j];
		}
		TempGroupNo[i] = 0;
		}
		/*******************************/

		/********************************
		//////////////�����������s�A���s�զX
		int RandomCPU;
		int RandomParticle1, RandomParticle2;
		Particle TempMemory;
		for (int j = 0; j<NumofGroup; j++)
		{
			for (int i = 0; i<NumofRedistribute; i++)
			{
				RandomCPU = RandomNumber(0, NumofGroup - 1);
				while (RandomCPU == j)
				{
					RandomCPU = RandomNumber(0, NumofGroup - 1);
				}
				RandomParticle1 = RandomNumber(0, NumofParticle - 1);
				RandomParticle2 = RandomNumber(0, NumofParticle - 1);
				//
				TempMemory = GroupInfo[j][RandomParticle1];
				GroupInfo[j][RandomParticle1] = GroupInfo[RandomCPU][RandomParticle2];
				GroupInfo[RandomCPU][RandomParticle2] = TempMemory;
			}
		}
		///////////*///��������

		/*�M�䭫���᪺�U�s��gbest*/
		for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
		{
			for (int i = 0; i<NumofParticle; i++)
			{
				GBestFitness[GroupNo] = ImpossibleResume;
			}
			for (int i = 0; i<NumofParticle; i++)
			{
				if (GBestFitness[GroupNo] > GroupInfo[GroupNo][i].GetPBestFitness() || GBestFitness[GroupNo] == ImpossibleResume)
				{
					GBest[GroupNo] = GroupInfo[GroupNo][i].GetPBest();
					GBestFitness[GroupNo] = GroupInfo[GroupNo][i].GetPBestFitness();
				}
			}
		}

		/****�M�䧹��****/

		for (int GroupNo = 0; GroupNo<NumofGroup; GroupNo++)
		{
			/*�U�s���s��X��gbest�g�^*/
			for(int i=0;i<NumofParticle;i++)
			{
			    GroupInfo[GroupNo][i].SetGBest(GBest[GroupNo],GBestFitness[GroupNo]);
			}
			/******/
			/*�N��X��Gbest�g�^���U�s*
			for (int i = 0; i<NumofLight; i++)
			{
				GBest[GroupNo][i] = TotalGBest[i];
			}
			GBestFitness[GroupNo] = TotalGBestFitness;
			/*************************/
		}

		/*�{���D�X���Ѥwí�w*
		if (RepeatValue == -1 && TotalGBestFitness != ImpossibleResume)
			RepeatValue = TotalGBestFitness;
		else if (RepeatValue - TotalGBestFitness < TotalGBestFitness * DifferentRange && RepeatValue != -1)
			break;
		/****/

	}
	EndTime = clock();//�����ɶ�


	/***PSO���浲���A�[�ݵ��G***/
	if (TotalGBestFitness != ImpossibleResume)
	{
		//cout<<"PSO�j�M"<<endl
		cout << "���O:";                                                                                 //fw<<"PSO�j�M"<<endl<<"���O:"; 
		for (int i = 0; i<NumofLight; i++)
		{
			cout << TotalGBest[i] << ",";                                                                            //fw<<GBest[i];  
		}
		cout << "�ӯ�:" << TotalGBestFitness << endl;                                                  //fw<<endl<<"�ӯ�:"<<GBest[NumofLight]<<endl;    
	}
	else
	{
		cout << "�L��" << endl;
	}
	cout << "�O��:" << double(EndTime - StartTime) / CLOCKS_PER_SEC << "��" << endl << endl;
	//fw.close();
	/*****************************/

	hipFree(d_GBest);
	hipFree(d_Group);
	hipFree(d_GBestFitness);

	hipFree(I);
	hipFree(Td);
	hipFree(KParameter);

	/*****write file*****/
	filename = "DPSOResult.txt";
	fw.open(filename, ios::out);//�}���ɮ�
	if (!fw){//�p�G�}���ɮץ��ѡAfw��0�F���\�Afw���D0
		cout << "Fail to open file: " << filename << endl;
	}
	if (TotalGBestFitness != ImpossibleResume)
	{
		fw << "���O:";                                                                                 //fw<<"PSO�j�M"<<endl<<"���O:"; 
		for (int i = 0; i<NumofLight; i++)
		{
			fw << TotalGBest[i] << ",";                                                                            //fw<<GBest[i];  
		}
		fw << "�ӯ�:" << TotalGBestFitness << endl;                                                  //fw<<endl<<"�ӯ�:"<<GBest[NumofLight]<<endl;    
	}
	else
	{
		fw << "�L��" << endl;
	}
	fw << "�O��:" << double(EndTime - StartTime) / CLOCKS_PER_SEC << "��" << endl << endl;
	fw.close();//�����ɮ�

	//����Ŷ�             //GroupInfozo �D�ʺA�}�C�A���Φۦ�R��
	//for (int i = 0; i < NumofGroup; i++)
	//{
	//	for (int j = 0; j < NumofParticle; j++)
	//	{
	//		delete[] GroupInfo;
	//	}
	//}
	//

	return TotalGBestFitness;
	//return (float(EndTime-StartTime)/CLOCKS_PER_SEC);
}



/****************************************Random Number**********************************************
�����Ʊ檺�̤j�ȻP�̤p�ȡA�N�ಣ�ͤ���o��ӭȤ����H���ơA�o���H���üƤ]�i�൥��̤j�ȡA�ε���̤p�ȡC
��üƭȬO�̷Ӯɶ��Ҳ��͡A�ҥH���ƩI�s�ɡA�|�������P���üƭ�(���)�C
***************************************************************************************************/
int RandomNumber(int MinValue, int MaxValue)   //Both MinValue and MaxValue are included
{
	int R = (rand() % (MaxValue - MinValue + 1)) + MinValue;
	return R;
}

/********************���ͰʺA2���}�C****************************/
void* new2d(int h, int w, int size)
{
	register int i;
	void **p;

	p = (void**)new char[h*sizeof(void*) + h*w*size];
	for (i = 0; i < h; i++)
	{
		p[i] = ((char *)(p + h)) + i*w*size;
	}

	return p;
}